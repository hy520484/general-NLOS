#include "hip/hip_runtime.h"
typedef float real;
// typedef double real;
typedef {data_type} data_type;	// np.int32 -> int, np.float32 -> float
#define c 3e8	// the speed of light

__global__ void bp(data_type* d_data, real* d_laserpoints, real* d_detectpoints, data_type* d_vol,
	float timeRes, int N_voxel_s, int N_voxel_t, int N_bin, float x_range0, float x_range1,
	float y_range0, float y_range1, float z_range0, float z_range1){
	// gridsize: (N_ld*N_ld, 1, 1); blocksize: (min(N_bin, 1024), 1, 1)
	// d_data: (N_ld, N_ld, N_bin)
	// d_laserpoints, d_detectpoints: (N_ld*N_ld, 2)
	// d_vol: (N_voxel_s, N_voxel_s, N_voxel_t)
	int bx = blockIdx.x;	// 0 ~ N_ld * N_ld - 1
	int tx = threadIdx.x;	// 0 ~ 1023
	// int tid = tx + bx * blockDim.x;	// not correct

	// one block deal with one pair of lp and dp, shares the following:
	__shared__ real s_lpx;
	__shared__ real s_lpy;
	__shared__ real s_lpz;
	__shared__ real s_dpx;
	__shared__ real s_dpy;
	__shared__ real s_dpz;
	extern __shared__ data_type s_data[];	// dynamic shared memory, length: N_bin

	// all blocks intialize d_vol to zeros, no need after python np.zeros() initialization
	// for (int k = 0; tid + blockDim.x * gridDim.x < N_voxel_s * N_voxel_s * N_voxel_t; k++){
	// 	d_vol[tid + blockDim.x * gridDim.x] = 0;
	// }

	// transport from d_data to s_data
	for (int k = 0; tx + k * blockDim.x < N_bin; k++){
		s_data[tx + k * blockDim.x] = d_data[tx + k * blockDim.x + bx * N_bin];
	}

	if (tx == 0){
		s_lpx = d_laserpoints[bx * 3 + 0];
		s_lpy = d_laserpoints[bx * 3 + 1];
		s_lpz = d_laserpoints[bx * 3 + 2];
		s_dpx = d_detectpoints[bx * 3 + 0];
		s_dpy = d_detectpoints[bx * 3 + 1];
		s_dpz = d_detectpoints[bx * 3 + 2];
	}
	__syncthreads();

	// backprojection, one thread deal with one voxel (vx, vy, vz)
	for (int k = 0; tx + k * blockDim.x < N_voxel_s * N_voxel_s * N_voxel_t; k++){
		int vx = (tx + k * blockDim.x) / N_voxel_s / N_voxel_t;
		int vy = (tx + k * blockDim.x - vx * N_voxel_s * N_voxel_t) / N_voxel_t;
		int vz = tx + k * blockDim.x - vx * N_voxel_s * N_voxel_t - vy * N_voxel_t;
		
		real x = x_range0 + (x_range1 - x_range0) / (N_voxel_s - 1) * real(vx);
		real y = y_range0 + (y_range1 - y_range0) / (N_voxel_s - 1) * real(vy);
		real z = z_range0 + (z_range1 - z_range0) / (N_voxel_t - 1) * real(vz);
		
		real t1 = sqrt((x - s_lpx) * (x - s_lpx) + (y - s_lpy) * (y - s_lpy) + (z - s_lpz) * (z - s_lpz));
		real t2 = sqrt((x - s_dpx) * (x - s_dpx) + (y - s_dpy) * (y - s_dpy) + (z - s_dpz) * (z - s_dpz));
		real d = t1 + t2;

		int bin = int(d / timeRes / c + 0.5);
		if (bin >= 10 && bin < N_bin){
			data_type num = s_data[bin];	// good
			// data_type num = data_type(float(s_data[bin]) * t1 * t1 * t2 * t2);	// bad
			// data_type num = data_type(float(s_data[bin]) * t1 * t2);	// good
			atomicAdd(&d_vol[tx + k * blockDim.x], num);
		}
	}
}
